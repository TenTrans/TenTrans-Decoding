#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "HUCudaHelper.h"
#include "HUNthElement.h"

namespace TenTrans{

#define UNROLL_MAXARG_LOOP(n, max)       \
  if(tid < (n) && tid + (n) < (max)) {   \
    if(sdata[tid + (n)] > sdata[tid]) {  \
      sdata[tid] = sdata[tid + (n)];     \
      indices[tid] = indices[tid + (n)]; \
    }                                    \
  }

template <typename T>
__global__ void gMaxElement(T* d_out, /* float* d_out*/
                            int* d_ind,
                            T* d_in,
                            int numBatches,
                            int* batchFirstElementIdxs) {
  // extern __shared__ float sdata[];
  extern __shared__ T sdata[];
  __shared__ int indices[512];

  int tid = threadIdx.x;

  for(int batchIdx = 0; batchIdx < numBatches; ++batchIdx) {
    int begin = batchFirstElementIdxs[batchIdx];
    int end = batchFirstElementIdxs[batchIdx + 1];

    int i = begin + blockIdx.x * (blockDim.x * 2) + tid;

    // sdata[tid] = -3.40282e+38f;
    sdata[tid] = (T)-10000;

    if(i < end) {
      // sdata[tid] = (float)d_in[i];
      sdata[tid] = d_in[i];
      indices[tid] = i;
    }

    if(i + blockDim.x < end) {
      // float a = (float)d_in[i];
      // float b = (float)d_in[i + blockDim.x];
      T a = d_in[i];
      T b = d_in[i + blockDim.x];
      if(a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while(i + 2 * gridDim.x * blockDim.x < end) {
      i += 2 * gridDim.x * blockDim.x;

      // float a = (float)d_in[i];
      T a = d_in[i];
      if(a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if(i + blockDim.x < end) {
        // float b = (float)d_in[i + blockDim.x];
        T b = d_in[i + blockDim.x];
        if(b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for(int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if(tid < s && tid + s < end) {
        if(sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, end);
    UNROLL_MAXARG_LOOP(16, end);
    UNROLL_MAXARG_LOOP(8, end);
    UNROLL_MAXARG_LOOP(4, end);
    UNROLL_MAXARG_LOOP(2, end);
    UNROLL_MAXARG_LOOP(1, end);

    if(tid == 0) {
      d_out[blockIdx.x + batchIdx * gridDim.x] = sdata[0];
      d_ind[blockIdx.x + batchIdx * gridDim.x] = indices[0];
    }
    __syncthreads();
  }
}

template <typename T>
__global__ void gMaxElementUpdate(T* binCosts, /* float* binCosts, */
                                  int* binIdxs,
                                  T* probs,
                                  int* batchFirstElements,
                                  T* outCosts, /* float* outCosts, */
                                  int* outIdxs,
                                  int* cummulatedBeamSizes,
                                  int NUM_BLOCKS) {
  // extern __shared__ float sdata[];
  extern __shared__ T sdata[];
  __shared__ int indices[512];
  // __shared__ float bestBinCost;
  __shared__ T bestBinCost;
  __shared__ int bestBinCostIdx;

  const int tid = threadIdx.x;
  const int batchIdx = blockIdx.x;
  const int N = batchFirstElements[batchIdx + 1] - batchFirstElements[batchIdx];
  int num_bins = int(N / (2 * 512)) + int(N % (2 * 512) != 0);
  if(num_bins > 500) {
    num_bins = 500;
  }

  for(int pos = cummulatedBeamSizes[batchIdx];
      pos < cummulatedBeamSizes[batchIdx + 1];
      ++pos) {
    int i = tid;

    // sdata[tid] = -3.40282e+38f;
    sdata[tid] = (T)-10000;

    if(i < num_bins) {
      sdata[tid] = binCosts[batchIdx * NUM_BLOCKS + i];
      indices[tid] = i;
    }

    if(i + blockDim.x < num_bins) {
      // float a = binCosts[batchIdx * NUM_BLOCKS + i];
      // float b = binCosts[batchIdx * NUM_BLOCKS + i + blockDim.x];
      T a = binCosts[batchIdx * NUM_BLOCKS + i];
      T b = binCosts[batchIdx * NUM_BLOCKS + i + blockDim.x];
      if(a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while(i + 2 * blockDim.x < num_bins) {
      i += 2 * blockDim.x;

      // float a = binCosts[batchIdx * NUM_BLOCKS + i];
      T a = binCosts[batchIdx * NUM_BLOCKS + i];
      if(a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if(i + blockDim.x < num_bins) {
        // float b = binCosts[batchIdx * NUM_BLOCKS + i + blockDim.x];
        T b = binCosts[batchIdx * NUM_BLOCKS + i + blockDim.x];
        if(b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for(int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if(tid < s && tid + s < num_bins) {
        if(sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, num_bins);
    UNROLL_MAXARG_LOOP(16, num_bins);
    UNROLL_MAXARG_LOOP(8, num_bins);
    UNROLL_MAXARG_LOOP(4, num_bins);
    UNROLL_MAXARG_LOOP(2, num_bins);
    UNROLL_MAXARG_LOOP(1, num_bins);

    if(tid == 0) {
      bestBinCost = sdata[0];
      bestBinCostIdx = batchIdx * NUM_BLOCKS + indices[0];

      // probs[binIdxs[bestBinCostIdx]] = (T)-3.40282e+38f;
      probs[binIdxs[bestBinCostIdx]] = (T) -10000;

      outIdxs[pos] = binIdxs[bestBinCostIdx];
      outCosts[pos] = bestBinCost;
    }

    __syncthreads();

    i = batchFirstElements[batchIdx]
        + (bestBinCostIdx - batchIdx * NUM_BLOCKS) * (blockDim.x * 2) + tid;
    const int dist = num_bins * 2 * blockDim.x;

    // sdata[tid] = -3.40282e+38f;
    sdata[tid] = (T)-10000;

    if(i < batchFirstElements[batchIdx + 1]) {
      // sdata[tid] = (float)probs[i];
      sdata[tid] = probs[i];
      indices[tid] = i;
    }

    if(i + blockDim.x < batchFirstElements[batchIdx + 1]) {
      // float a = (float)probs[i];
      // float b = (float)probs[i + blockDim.x];
      T a = probs[i];
      T b = probs[i + blockDim.x];
      if(a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while(i + dist < batchFirstElements[batchIdx + 1]) {
      i += dist;

      // float a = (float)probs[i];
      T a = probs[i];
      if(a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if(i + blockDim.x < batchFirstElements[batchIdx + 1]) {
        // float b = (float)probs[i + blockDim.x];
        T b = probs[i + blockDim.x];
        if(b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for(int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if(tid < s && tid + s < batchFirstElements[batchIdx + 1]) {
        if(sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(16, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(8, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(4, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(2, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(1, batchFirstElements[batchIdx + 1]);

    if(tid == 0) {
      binCosts[bestBinCostIdx] = sdata[0];
      binIdxs[bestBinCostIdx] = indices[0];
    }
    __syncthreads();
  }
}

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < n) {
    int index = indeces[tid];
    d_out[tid] = d_in[index];
  }
}

class NthElementGPU {
public:
  NthElementGPU() = delete;
  NthElementGPU(const NthElementGPU& copy) = delete;

  NthElementGPU(size_t maxBeamSize,
                size_t maxBatchSize,
                DeviceId deviceId)
      : deviceId_(deviceId),
        NUM_BLOCKS(std::min(
            500,
            int(maxBeamSize* MAX_VOCAB_SIZE / (2 * BLOCK_SIZE))
                + int(maxBeamSize* MAX_VOCAB_SIZE % (2 * BLOCK_SIZE) != 0))) {
    // std::cerr << "NthElement::NthElement" << std::endl;

    hipSetDevice(deviceId_.no);

    CUDA_CHECK(hipMalloc((void**)&d_ind, maxBatchSize * NUM_BLOCKS * sizeof(int)));
    // CUDA_CHECK(hipMalloc((void**)&d_out, maxBatchSize * NUM_BLOCKS * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_out, maxBatchSize * NUM_BLOCKS * sizeof(TT_DATA_TYPE)));

    CUDA_CHECK(hipMalloc((void**)&d_res_idx, maxBatchSize * maxBeamSize * sizeof(int)));
    // CUDA_CHECK(hipMalloc((void**)&d_res, maxBatchSize * maxBeamSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_res, maxBatchSize * maxBeamSize * sizeof(TT_DATA_TYPE)));

    // CUDA_CHECK(hipHostAlloc((void**)&h_res, maxBeamSize * maxBatchSize * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_res, maxBeamSize * maxBatchSize * sizeof(TT_DATA_TYPE), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_res_idx, maxBeamSize * maxBatchSize * sizeof(int), hipHostMallocDefault));

    CUDA_CHECK(hipMalloc((void**)&d_breakdown, maxBeamSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_batchPosition, (maxBatchSize + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_cumBeamSizes,  (maxBatchSize + 1) * sizeof(int)));
  }

  ~NthElementGPU() {
    hipSetDevice(deviceId_.no);

    CUDA_CHECK(hipFree(d_cumBeamSizes));
    CUDA_CHECK(hipFree(d_batchPosition));
    CUDA_CHECK(hipFree(d_breakdown));
    CUDA_CHECK(hipHostFree(h_res_idx));
    CUDA_CHECK(hipHostFree(h_res));
    CUDA_CHECK(hipFree(d_res));
    CUDA_CHECK(hipFree(d_res_idx));
    CUDA_CHECK(hipFree(d_out));
    CUDA_CHECK(hipFree(d_ind));
  }

private:
  void getNBestList(TT_DATA_TYPE* probs,
                    const std::vector<int>& batchFirstElementIdxs,
                    const std::vector<int>& cummulatedBeamSizes) 
  {
    hipSetDevice(deviceId_.no);
    CUDA_CHECK(hipMemcpyAsync(d_batchPosition,
                               batchFirstElementIdxs.data(),
                               batchFirstElementIdxs.size() * sizeof(int),
                               hipMemcpyHostToDevice,
                               /* stream_ */ 0));
    CUDA_CHECK(hipMemcpyAsync(d_cumBeamSizes,
                               cummulatedBeamSizes.data(),
                               cummulatedBeamSizes.size() * sizeof(int),
                               hipMemcpyHostToDevice,
                               /* stream_ */ 0));

    const int numBatches = batchFirstElementIdxs.size() - 1;

    gMaxElement<TT_DATA_TYPE><<<NUM_BLOCKS, 
                                BLOCK_SIZE, 
                                BLOCK_SIZE * sizeof(TT_DATA_TYPE), 
                                /* stream_ */ 0>>>
                                (d_out, d_ind, probs, numBatches, d_batchPosition);

    gMaxElementUpdate<TT_DATA_TYPE><<<numBatches,
                                      BLOCK_SIZE,
                                      BLOCK_SIZE * sizeof(TT_DATA_TYPE),
                                      /* stream_ */ 0>>>
                                      (d_out, d_ind, probs, d_batchPosition, d_res, d_res_idx, d_cumBeamSizes, NUM_BLOCKS);
  }

public:
  void getNBestList(const std::vector<size_t>& beamSizes,
                    HUPtr<HUTensor> Probs,
                    std::vector<float>& outCosts,
                    std::vector<unsigned>& outKeys,
                    const bool isFirst) {
    hipSetDevice(deviceId_.no);

    std::vector<int> cummulatedBeamSizes(beamSizes.size() + 1, 0);
    std::vector<int> batchFirstElementIdxs(beamSizes.size() + 1, 0);

    const size_t vocabSize = Probs->shape()[-1];

    for(size_t i = 0; i < beamSizes.size(); ++i) {
      cummulatedBeamSizes[i + 1] = cummulatedBeamSizes[i] + beamSizes[i];
      batchFirstElementIdxs[i + 1]
          += ((isFirst) ? (i + 1) : cummulatedBeamSizes[i + 1]) * vocabSize;
    }

    getNBestList(Probs->data(), batchFirstElementIdxs, cummulatedBeamSizes);
    getPairs(cummulatedBeamSizes.back(), outKeys, outCosts);
  }

private:
  void getPairs(size_t number,
                std::vector<unsigned>& outKeys,
                std::vector<float>& outValues) {
    hipSetDevice(deviceId_.no);

    CUDA_CHECK(hipMemcpyAsync(h_res,
                               d_res,
                               number * sizeof(TT_DATA_TYPE),
                               hipMemcpyDeviceToHost,
                               /* stream_ */ 0));
    CUDA_CHECK(hipMemcpyAsync(h_res_idx,
                               d_res_idx,
                               number * sizeof(int),
                               hipMemcpyDeviceToHost,
                               /* stream_ */ 0));
    hipStreamSynchronize(/* stream_ */ 0);

    for(size_t i = 0; i < number; ++i) {
      outKeys.push_back(h_res_idx[i]);
      outValues.push_back((float)h_res[i]);
    }

    lastN = number;
  }

  void getValueByKey(std::vector<float>& out, float* d_in) {
    hipSetDevice(deviceId_.no);

    gGetValueByKey<<<1, lastN, 0, /* stream_ */ 0>>>(
        d_in, d_breakdown, h_res_idx, lastN);

    CUDA_CHECK(hipMemcpyAsync(out.data(),
                               d_breakdown,
                               lastN * sizeof(float),
                               hipMemcpyDeviceToHost,
                               /* stream_ */ 0));
    CUDA_CHECK(hipStreamSynchronize(/* stream_ */ 0));
  }

  DeviceId deviceId_;

  const int MAX_VOCAB_SIZE = 100000;

  const int BLOCK_SIZE = 512;
  const int NUM_BLOCKS;

  int* d_ind;
  // float* d_out;
  TT_DATA_TYPE* d_out;

  int* d_res_idx;
  // float* d_res;
  TT_DATA_TYPE* d_res;

  int* h_res_idx;
  // float* h_res;
  TT_DATA_TYPE* h_res;

  float* d_breakdown;
  int* d_batchPosition;
  int* d_cumBeamSizes;
  size_t lastN;
};

GetNBestListFn createGetNBestListFn(size_t beamSize, size_t dimBatch, DeviceId deviceId) 
{
  auto nth = HUNew<NthElementGPU>(beamSize, dimBatch, deviceId);
  return [nth](const std::vector<size_t>& beamSizes,
      HUPtr<HUTensor> logProbs,
      std::vector<float>& outCosts,
      std::vector<unsigned>& outKeys,
      const bool isFirst) {
      return nth->getNBestList(beamSizes, logProbs, outCosts, outKeys, isFirst);
  };
}

}
